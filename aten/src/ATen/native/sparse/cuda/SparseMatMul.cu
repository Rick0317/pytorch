#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/Config.h>
#include <ATen/Dispatch.h>
#include <ATen/NamedTensorUtils.h>
#include <ATen/Parallel.h>
#include <ATen/SparseTensorImpl.h>
#include <ATen/SparseTensorUtils.h>
#include <ATen/native/Resize.h>
#include <hip/hip_runtime.h>
#include <type_traits>

#include <cusparseLt.h>


#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_sparse_sparse_matmul_native.h>
#include <ATen/ops/empty.h>
#include <ATen/ops/empty_like_native.h>
#endif

#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <thrust/sequence.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDADataType.h>
#include <ATen/cuda/CUDAUtils.h>
#include <ATen/cuda/ThrustAllocator.h>
#include <hipsparse.h>
#include <ATen/native/sparse/cuda/SparseCUDABlas.h>
#include <c10/cuda/CUDACachingAllocator.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/discard_iterator.h>


#if defined(__HIPCC__) && (CUSPARSE_VERSION >= 11000)
#define IS_CUSPARSE11_AVAILABLE() 1
#else
#define IS_CUSPARSE11_AVAILABLE() 0
#endif

#if IS_CUSPARSE11_AVAILABLE()
#include <hip/library_types.h>
#endif


#define CHECK_CUDA(func)                                      \
  {                                                           \
    hipError_t status = (func);                              \
    if (status != hipSuccess) {                              \
      printf(                                                 \
          "CUDA API failed at line %d with error: %s (%d)\n", \
          __LINE__,                                           \
          hipGetErrorString(status),                         \
          status);                                            \
      return at::Tensor{};                                    \
    }                                                         \
  }

#define CHECK_CUSPARSE(func)                                      \
  {                                                               \
    hipsparseStatus_t status = (func);                             \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                      \
      printf(                                                     \
          "CUSPARSE API failed at line %d with error: %s (%d)\n", \
          __LINE__,                                               \
          hipsparseGetErrorString(status),                         \
          status);                                                \
      return at::Tensor{};                                        \
    }                                                             \
  }


constexpr int EXIT_UNSUPPORTED = 2;
constexpr float EPSILON = 0.01;


namespace at {
namespace native {

at::Tensor _cusparselt_masked_mm(
  const at::Tensor& A, const at::Tensor& B, const at::Tensor& C,
  int64_t m, int64_t n, int64_t k, int64_t iters,
  int64_t gpu_index, int64_t check_correctness,
  int64_t endtoend)
{
  CHECK_CUDA(hipSetDevice(gpu_index));

  int major_cc, minor_cc;
  CHECK_CUDA(
      hipDeviceGetAttribute(&major_cc, hipDeviceAttributeComputeCapabilityMajor, 0))
  CHECK_CUDA(
      hipDeviceGetAttribute(&minor_cc, hipDeviceAttributeComputeCapabilityMinor, 0))
  if (!(major_cc == 8 && minor_cc == 0)) {
    std::printf(
        "\ncusparseLt is supported only on GPU devices with"
        " compute capability == 8.0, current: %d.%d\n\n",
        major_cc,
        minor_cc);
    return at::Tensor{};
  }

  auto order = HIPSPARSE_ORDER_ROW;
  auto opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  auto opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  auto type = HIP_R_16F;
  auto compute_type = CUSPARSE_COMPUTE_16F;

  if (sizeof(float) == 4) {
    type = HIP_R_32F;
    compute_type = CUSPARSE_COMPUTE_TF32_FAST;
  } else if (sizeof(float) == 1) {
    type = HIP_R_8I;
    compute_type = CUSPARSE_COMPUTE_32I;
    opB = HIPSPARSE_OPERATION_TRANSPOSE;
  }

  bool is_rowmajor = (order == HIPSPARSE_ORDER_ROW);
  bool isA_transposed = (opA != HIPSPARSE_OPERATION_NON_TRANSPOSE);
  bool isB_transposed = (opB != HIPSPARSE_OPERATION_NON_TRANSPOSE);
  auto num_A_rows = A.size(0);
  auto num_A_cols = A.size(1);
  auto num_B_rows = B.size(0);
  auto num_B_cols = B.size(1);
  auto num_C_rows = A.size(0);
  auto num_C_cols = B.size(1);
  unsigned alignment = 16;
  auto lda = (is_rowmajor) ? num_A_cols : num_A_rows;
  auto ldb = (is_rowmajor) ? num_B_cols : num_B_rows;
  auto ldc = (is_rowmajor) ? num_C_cols : num_C_rows;
  auto A_height = (is_rowmajor) ? num_A_rows : num_A_cols;
  auto B_height = (is_rowmajor) ? num_B_rows : num_B_cols;
  auto C_height = (is_rowmajor) ? num_C_rows : num_C_cols;
  auto A_size = A_height * lda * sizeof(float);
  auto B_size = B_height * ldb * sizeof(float);
  auto C_size = C_height * ldc * sizeof(float);
  auto hA = A.data_ptr();
  auto hB = B.data_ptr();
  auto hC = C.data_ptr();
  // T *hA, *hB, *hC;
  // CHECK_CUDA(hipHostMalloc((void**)&hA, A_size));
  // CHECK_CUDA(hipHostMalloc((void**)&hB, B_size));
  // CHECK_CUDA(hipHostMalloc((void**)&hC, C_size));
  float alpha = 1.0f;
  float beta = 0.0f;

  //--------------------------------------------------------------------------
  // Device memory management
  float *dA, *dB, *dC, *dD, *dA_compressed;
  CHECK_CUDA(hipMalloc((void**)&dA, A_size))
  CHECK_CUDA(hipMalloc((void**)&dB, B_size))
  CHECK_CUDA(hipMalloc((void**)&dC, C_size))
  dD = dC;

  CHECK_CUDA(hipMemcpy(dA, hA, A_size, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(dB, hB, B_size, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(dC, hC, C_size, hipMemcpyHostToDevice))
  //--------------------------------------------------------------------------
  cusparseLtHandle_t handle;
  cusparseLtMatDescriptor_t matA, matB, matC;
  cusparseLtMatmulDescriptor_t matmul;
  cusparseLtMatmulAlgSelection_t alg_sel;
  cusparseLtMatmulPlan_t plan;
  hipStream_t stream = nullptr;
  CHECK_CUSPARSE(cusparseLtInit(&handle))
  // matrix descriptor initilization
  CHECK_CUSPARSE(cusparseLtStructuredDescriptorInit(
      &handle,
      &matA,
      num_A_rows,
      num_A_cols,
      lda,
      alignment,
      type,
      order,
      CUSPARSELT_SPARSITY_50_PERCENT))
  CHECK_CUSPARSE(cusparseLtDenseDescriptorInit(
      &handle, &matB, num_B_rows, num_B_cols, ldb, alignment, type, order))
  CHECK_CUSPARSE(cusparseLtDenseDescriptorInit(
      &handle, &matC, num_C_rows, num_C_cols, ldc, alignment, type, order))
  // matmul, algorithm selection, and plan initilization
  CHECK_CUSPARSE(cusparseLtMatmulDescriptorInit(
      &handle, &matmul, opA, opB, &matA, &matB, &matC, &matC, compute_type))
  CHECK_CUSPARSE(cusparseLtMatmulAlgSelectionInit(
      &handle, &alg_sel, &matmul, CUSPARSELT_MATMUL_ALG_DEFAULT))
  int alg = 0;
  CHECK_CUSPARSE(cusparseLtMatmulAlgSetAttribute(
      &handle, &alg_sel, CUSPARSELT_MATMUL_ALG_CONFIG_ID, &alg, sizeof(alg)))
  size_t workspace_size, compressed_size;
  CHECK_CUSPARSE(
      cusparseLtMatmulGetWorkspace(&handle, &plan, &workspace_size))

  CHECK_CUSPARSE(cusparseLtMatmulPlanInit(
      &handle, &plan, &matmul, &alg_sel, workspace_size))
  //--------------------------------------------------------------------------
  // Prune the A matrix (in-place) and check the correcteness
  CHECK_CUSPARSE(cusparseLtSpMMAPrune2(
      &handle, &matA, 1, opA, dA, dA, CUSPARSELT_PRUNE_SPMMA_TILE, stream))

  int *is_valid;
  CHECK_CUDA(hipMalloc((void**)&is_valid, sizeof(int)))
  CHECK_CUSPARSE(
      cusparseLtSpMMAPruneCheck2(&handle, &matA, 1, opA, dA, is_valid, stream))
  int h_is_valid = 0;
  CHECK_CUDA(hipMemcpy(&h_is_valid, is_valid, sizeof(int), hipMemcpyDeviceToHost))
  CHECK_CUDA(hipFree(is_valid))

  if (h_is_valid != 0) {
    std::printf(
        "!!!! The matrix has been pruned in a wrong way. "
        "cusparseLtMatmul will not provided correct results\n");
    return at::Tensor{};
  }

  // Measure time with CUDA events
  hipEvent_t t_start, t_stop;
  CHECK_CUDA(hipEventCreate(&t_start));
  CHECK_CUDA(hipEventCreate(&t_stop));

  float t_min_ms = 1e+10f;
  float t_max_ms = 0.0f;
  float t_avg_ms = 0.0f;
  float t_cur_ms = 0.0f;

  //--------------------------------------------------------------------------
  // Compress the A matrix
  CHECK_CUSPARSE(
      cusparseLtSpMMACompressedSize2(&handle, &matA, &compressed_size))
  CHECK_CUDA(hipMalloc((void**)&dA_compressed, compressed_size))

  CHECK_CUSPARSE(
    cusparseLtSpMMACompress2(&handle, &matA, 1, opA, dA, dA_compressed, stream))

  //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
  // Perform the matrix multiplication
  void* d_workspace = nullptr;
  int num_streams = 0;
  hipStream_t* streams = nullptr;

  // Warmup
  CHECK_CUSPARSE(cusparseLtMatmul(
      &handle,
      &plan,
      &alpha,
      dA_compressed,
      dB,
      &beta,
      dC,
      dD,
      d_workspace,
      streams,
      num_streams))

  for (int i = 0; i < iters; ++i) {
    hipEventRecord(t_start);

    if (endtoend) {
      CHECK_CUSPARSE(
        cusparseLtSpMMACompress2(&handle, &matA, 1, opA, dA, dA_compressed, stream))
    }

    CHECK_CUSPARSE(cusparseLtMatmul(
        &handle,
        &plan,
        &alpha,
        dA_compressed,
        dB,
        &beta,
        dC,
        dD,
        d_workspace,
        streams,
        num_streams))

    hipEventRecord(t_stop);
    hipEventSynchronize(t_stop);
    hipEventElapsedTime(&t_cur_ms, t_start, t_stop);

    t_min_ms = (t_cur_ms <= t_min_ms) ? t_cur_ms : t_min_ms;
    t_max_ms = (t_cur_ms >= t_max_ms) ? t_cur_ms : t_max_ms;
    t_avg_ms += t_cur_ms;
  }
  t_avg_ms /= (float)iters;

  // Print effective GFLOP/s
  double num_gflop = (double)m * double(n) * double(k) * 2.0 / 1e+9;
  double max_gflops = num_gflop / (double)t_min_ms * 1e+3;
  double min_gflops = num_gflop / (double)t_max_ms * 1e+3;
  double avg_gflops = num_gflop / (double)t_avg_ms * 1e+3;
  std::cout << m << "," << n << "," << k << ",";
  std::cout << min_gflops << "," << max_gflops << "," << avg_gflops << std::endl;

  // Help dump the profiler stats
  hipDeviceSynchronize();

  //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
  // destroy plan and handle
  CHECK_CUSPARSE(cusparseLtMatmulPlanDestroy(&plan))
  CHECK_CUSPARSE(cusparseLtDestroy(&handle))
  //--------------------------------------------------------------------------
  // device result check
  // matrix A has been pruned
  CHECK_CUDA(hipMemcpy(hA, dA, A_size, hipMemcpyDeviceToHost))
  CHECK_CUDA(hipMemcpy(hC, dC, C_size, hipMemcpyDeviceToHost))

  // device memory deallocation
  CHECK_CUDA(hipFree(dA_compressed))
  CHECK_CUDA(hipFree(dA))
  CHECK_CUDA(hipFree(dB))
  CHECK_CUDA(hipFree(dC))
  CHECK_CUDA(hipHostFree(hA))
  CHECK_CUDA(hipHostFree(hB))
  CHECK_CUDA(hipHostFree(hC))

  return C;
}

} // at
} // native


namespace at {
namespace native {

namespace {

using namespace at::sparse;

Tensor _to_csr_int(const Tensor& rowIndices, int64_t dim, int64_t nnz) {
  Tensor csr = at::empty({dim + 1}, CUDA(kInt));
  Tensor rowIndicesInt = at::empty({rowIndices.size(0)}, CUDA(kInt));
  rowIndicesInt.copy_(rowIndices);
  sparse::cuda::Xcoo2csr(
      rowIndicesInt.data_ptr<int32_t>(), nnz, dim, csr.data_ptr<int32_t>());
  return csr;
}


#pragma push
// NVCC complains that confirm_mult_size is not used,
// but it is used in specializations of CusparseMatrixMultiplyOp below
#pragma diag_suppress 177   // Function was declared but never referenced
int confirm_mult_size(const std::vector<int>& mat1_size, const std::vector<int>& mat2_size) {
  TORCH_CHECK(
      mat1_size[1] == mat2_size[0],
      "mat1 and mat2 shapes cannot be multiplied (",
      mat1_size[0],
      "x",
      mat1_size[1],
      " and ",
      mat2_size[0],
      "x",
      mat2_size[1],
      ")");
  return mat1_size[1];
}
#pragma pop

void create_general_description_(hipsparseMatDescr_t& description_) {
  TORCH_CUDASPARSE_CHECK(hipsparseCreateMatDescr(&description_));
  TORCH_CUDASPARSE_CHECK(hipsparseSetMatType(description_, HIPSPARSE_MATRIX_TYPE_GENERAL));
  TORCH_CUDASPARSE_CHECK(hipsparseSetMatIndexBase(description_, HIPSPARSE_INDEX_BASE_ZERO));
}

// csrMatrixRef is used to have a representation of a raw CSR matrix representation
// comming from `sparse_sparse_matmul_cuda_kernel` function.
// Moreover this implements a RAII guard for a cusparse descriptor
template<class scalar_t>
struct csrMatrixRef {
  int* csr_indices_{nullptr};
  int* csr_pointers_{nullptr};
  scalar_t* csr_values_{nullptr};
  int nnz_{0};
  std::vector<int> size_{};

  #if IS_CUSPARSE11_AVAILABLE()
    hipsparseSpMatDescr_t description_{0};
  #else
    hipsparseMatDescr_t description_{0};
  #endif

  csrMatrixRef() {
    #if !IS_CUSPARSE11_AVAILABLE()
      create_general_description_(description_);
    #endif
  }

  csrMatrixRef(
      int* csr_indices,
      int* csr_pointers,
      scalar_t* csr_values,
      int nnz,
      const std::vector<int>& size)
      : csr_indices_{csr_indices},
        csr_pointers_{csr_pointers},
        csr_values_{csr_values},
        nnz_{nnz},
        size_{size} {
    #if IS_CUSPARSE11_AVAILABLE()
      hipDataType cuda_data_type = at::cuda::getCudaDataType<scalar_t>();
      TORCH_CUDASPARSE_CHECK(hipsparseCreateCsr(
        &description_,
        this->size(0),
        this->size(1),
        this->nnz_,
        this->csr_pointers_,
        this->csr_indices_,
        this->csr_values_,
        HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO,
        cuda_data_type));
    #else
      create_general_description_(description_);
    #endif
  }

  ~csrMatrixRef() {
    #if IS_CUSPARSE11_AVAILABLE()
      hipsparseDestroySpMat(description_);
    #else
      hipsparseDestroyMatDescr(description_);
    #endif
  }

  int size(int index) const {
    return size_.at(index);
  }
};

// csrOutput is used to represent the output for `CusparseMatrixMultiplyOp`
// Note that `csrOutput` is different from `csrMatrixRef` and the purpose
// of this was to have a materialized  version of a CSR matrix.
// Moreover this implements a RAII guard for a cusparse descriptor
struct csrOutput {
  Tensor csr_indices_{};
  Tensor csr_pointers_{};
  at::Tensor csr_values_{};
  int nnz_{0};
  std::vector<int> size_;

  hipsparseMatDescr_t description_{0};

  csrOutput(const std::vector<int> &size) : size_{size} {
    create_general_description_(description_);
  }

  ~csrOutput() {
    hipsparseDestroyMatDescr(description_);
  }

  int size(int index) const {
    return size_.at(index);
  }
};

#if IS_CUSPARSE11_AVAILABLE()

// RAII guard helps to support cuSparse 11 API for `A @ B` operation
// This generic template exists because with cuSparse the `scalar_t` type could be a double or float
template <class scalar_t>
struct CusparseMatrixMultiplyOp {

  hipsparseSpGEMMDescr_t spgemmDesc;

  CusparseMatrixMultiplyOp() {
    static_assert(
      std::is_same<c10::Half, scalar_t>::value ||
          std::is_same<c10::BFloat16, scalar_t>::value ||
          std::is_same<float, scalar_t>::value ||
          std::is_same<double, scalar_t>::value ||
          std::is_same<c10::complex<float>, scalar_t>::value ||
          std::is_same<c10::complex<double>, scalar_t>::value,
      "cusparseSpGEMM only supports data type of half, bfloat16, float, double and complex float, double.");
    // SpGEMM Computation
    TORCH_CUDASPARSE_CHECK(hipsparseSpGEMM_createDescr(&spgemmDesc));
  }

  ~CusparseMatrixMultiplyOp() {
    // destroy matrix/vector descriptors
    hipsparseSpGEMM_destroyDescr(spgemmDesc);
  }

  csrOutput operator ()(
      const csrMatrixRef<scalar_t>& A,
      const csrMatrixRef<scalar_t>& B,
      Tensor& output_values,
      Tensor& output_indices) {
    const int A_num_rows = A.size(0);

    const int B_num_cols = B.size(1);

    csrOutput out({A.size(0), B.size(1)});

    out.csr_pointers_ = at::empty({out.size(0) + 1}, output_indices.options().dtype(kInt));

    int* dC_csrOffsets = out.csr_pointers_.data_ptr<int>();
    int* dC_columns = nullptr;
    scalar_t* dC_values = nullptr;

    scalar_t alpha = 1.0f;
    scalar_t beta = 0.0f;
    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;

    csrMatrixRef<scalar_t> C(
      nullptr,
      nullptr,
      nullptr,
      /*nnz*/0,
      {A_num_rows, B_num_cols}
    );

    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t handle = at::cuda::getCurrentCUDASparseHandle();
    void *dBuffer1 = NULL, *dBuffer2 = NULL;
    size_t bufferSize1 = 0, bufferSize2 = 0;

    hipsparseSpMatDescr_t matA = A.description_;
    hipsparseSpMatDescr_t matB = B.description_;
    hipsparseSpMatDescr_t matC = C.description_;
    //--------------------------------------------------------------------------

    hipDataType computeType = at::cuda::getCudaDataType<scalar_t>();

    // If a specific GPU model does not provide native support for a given data type,
    // the routine returns HIPSPARSE_STATUS_ARCH_MISMATCH error
    hipDeviceProp_t* prop = at::cuda::getCurrentDeviceProperties();
    TORCH_CHECK(prop->major >= 5 && !((10*prop->major + prop->minor) < 53 && computeType == HIP_R_16F),
        "sparse_mm: CUDA Float16 requires compute capability >= 53 (current: ", prop->major, prop->minor, ")");
    TORCH_CHECK(!(prop->major < 8 && computeType == HIP_R_16BF),
        "sparse_mm: CUDA BFloat16 requires compute capability >= 80 (current: ", prop->major, prop->minor, ")");

    // ask bufferSize1 bytes for external memory
    TORCH_CUDASPARSE_CHECK(hipsparseSpGEMM_workEstimation(
        handle,
        opA,
        opB,
        &alpha,
        matA,
        matB,
        &beta,
        matC,
        computeType,
        HIPSPARSE_SPGEMM_DEFAULT,
        spgemmDesc,
        &bufferSize1,
        NULL));

    auto& allocator = *::c10::cuda::CUDACachingAllocator::get();

    at::DataPtr dataPtr1 = allocator.allocate(bufferSize1);
    dBuffer1 = dataPtr1.get();
    // inspect the matrices A and B to understand the memory requiremnent for
    // the next step
    TORCH_CUDASPARSE_CHECK(hipsparseSpGEMM_workEstimation(
        handle,
        opA,
        opB,
        &alpha,
        matA,
        matB,
        &beta,
        matC,
        computeType,
        HIPSPARSE_SPGEMM_DEFAULT,
        spgemmDesc,
        &bufferSize1,
        dBuffer1));

    // ask bufferSize2 bytes for external memory
    TORCH_CUDASPARSE_CHECK(hipsparseSpGEMM_compute(
        handle,
        opA,
        opB,
        &alpha,
        matA,
        matB,
        &beta,
        matC,
        computeType,
        HIPSPARSE_SPGEMM_DEFAULT,
        spgemmDesc,
        &bufferSize2,
        NULL));

    at::DataPtr dataPtr2 = allocator.allocate(bufferSize2);
    dBuffer2 = dataPtr2.get();

    // compute the intermediate product of A * B
    TORCH_CUDASPARSE_CHECK(hipsparseSpGEMM_compute(
        handle,
        opA,
        opB,
        &alpha,
        matA,
        matB,
        &beta,
        matC,
        computeType,
        HIPSPARSE_SPGEMM_DEFAULT,
        spgemmDesc,
        &bufferSize2,
        dBuffer2));
    // get matrix C non-zero entries C_num_nnz1
    int64_t C_num_rows1, C_num_cols1, C_num_nnz1;
    TORCH_CUDASPARSE_CHECK(
        hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1, &C_num_nnz1));
    // allocate matrix C
    // allocate C offsets
    out.nnz_ = C_num_nnz1;

    out.csr_indices_ = at::empty({out.nnz_}, output_indices.options().dtype(kInt));
    out.csr_values_ = at::empty({out.nnz_}, output_values.options());
    dC_columns = out.csr_indices_.data_ptr<int>();
    dC_values = out.csr_values_.data_ptr<scalar_t>();

    // update matC with the new pointers
    TORCH_CUDASPARSE_CHECK(
        hipsparseCsrSetPointers(matC, dC_csrOffsets, dC_columns, dC_values));

    // copy the final products to the matrix C
    TORCH_CUDASPARSE_CHECK(hipsparseSpGEMM_copy(
        handle,
        opA,
        opB,
        &alpha,
        matA,
        matB,
        &beta,
        matC,
        computeType,
        HIPSPARSE_SPGEMM_DEFAULT,
        spgemmDesc));
    return out;
  }
};


template struct CusparseMatrixMultiplyOp<float>;

template struct CusparseMatrixMultiplyOp<double>;

#else // if not IS_CUSPARSE11_AVAILABLE()

using DcsrMatrixRef = csrMatrixRef<double>;
using ScsrMatrixRef = csrMatrixRef<float>;

// RAII guard helps to support cuSparse 10 API for `A @ B` operation
// This generic template exists because with cuSparse the `scalar_t` type could be a double or float
template <class scalar_t>
struct CusparseMatrixMultiplyOp {
  csrOutput operator()(
      const csrMatrixRef<scalar_t>& lhs,
      const csrMatrixRef<scalar_t>& rhs,
      Tensor &output_values,
      Tensor &output_indices)
  {
    TORCH_INTERNAL_ASSERT(false, "cusparse csr sparse-sparse MM only supports data type of float and double.");
  }
};

// Specializacion for `A @ B` operation for double values with cuSparse
template<> struct CusparseMatrixMultiplyOp<double> {
  csrgemm2Info_t gemm2Info_;

  CusparseMatrixMultiplyOp() {
    TORCH_CUDASPARSE_CHECK(hipsparseCreateCsrgemm2Info(&gemm2Info_));
  }
  ~CusparseMatrixMultiplyOp() {
    hipsparseDestroyCsrgemm2Info(gemm2Info_);
  }

  csrOutput operator ()(
      const DcsrMatrixRef& lhs,
      const DcsrMatrixRef& rhs,
      Tensor &output_values,
      Tensor &output_indices) {
    double alpha = 1.0;
    DcsrMatrixRef empty;
    return Dgemm2(lhs, rhs, empty, &alpha, nullptr, output_values, output_indices);
  }

  csrOutput Dgemm2(
      const DcsrMatrixRef& A,
      const DcsrMatrixRef& B,
      const DcsrMatrixRef& C,
      const double* alpha,
      const double* beta,
      Tensor &output_values,
      Tensor &output_indices) {
    void* buffer_{nullptr};
    hipsparseHandle_t cusparseHandle_ = at::cuda::getCurrentCUDASparseHandle();
    TORCH_CUDASPARSE_CHECK(hipsparseSetPointerMode(cusparseHandle_, HIPSPARSE_POINTER_MODE_HOST));

    csrOutput out({A.size(0), B.size(1)});
    int innerSize = confirm_mult_size(A.size_, B.size_);
    out.csr_pointers_ = at::empty({out.size(0) + 1}, output_indices.options().dtype(kInt));

    // Compute needed buffer size
    size_t new_bubber_sz;
    TORCH_CUDASPARSE_CHECK(hipsparseDcsrgemm2_bufferSizeExt(
        cusparseHandle_,
        out.size(0),
        out.size(1),
        innerSize,
        alpha,
        A.description_,
        A.nnz_,
        A.csr_pointers_,
        A.csr_indices_,
        B.description_,
        B.nnz_,
        B.csr_pointers_,
        B.csr_indices_,
        beta,
        C.description_,
        C.nnz_,
        C.csr_pointers_,
        C.csr_indices_,
        gemm2Info_,
        &new_bubber_sz));

    // (Re)allocate buffer if needed
    auto& allocator = *::c10::cuda::CUDACachingAllocator::get();
    at::DataPtr data_ptr = allocator.allocate(new_bubber_sz);
    buffer_ = data_ptr.get();

    // Find the resulting non-zero pattern.
    TORCH_CUDASPARSE_CHECK(hipsparseXcsrgemm2Nnz(
        cusparseHandle_,
        out.size(0),
        out.size(1),
        innerSize,
        A.description_,
        A.nnz_,
        A.csr_pointers_,
        A.csr_indices_,
        B.description_,
        B.nnz_,
        B.csr_pointers_,
        B.csr_indices_,
        C.description_,
        C.nnz_,
        C.csr_pointers_,
        C.csr_indices_,
        out.description_,
        out.csr_pointers_.data_ptr<int>(),
        &out.nnz_,
        gemm2Info_,
        buffer_));

    out.csr_indices_ = at::empty({out.nnz_}, output_indices.options().dtype(kInt));
    out.csr_values_ = at::empty({out.nnz_}, output_values.options());

    // Perform the gemm2 operation for doubles
    // out = alpha ∗ A ∗ B + beta ∗ C
    TORCH_CUDASPARSE_CHECK(hipsparseDcsrgemm2(
        cusparseHandle_,
        out.size(0),
        out.size(1),
        innerSize,
        alpha,
        A.description_,
        A.nnz_,
        A.csr_values_,
        A.csr_pointers_,
        A.csr_indices_,
        B.description_,
        B.nnz_,
        B.csr_values_,
        B.csr_pointers_,
        B.csr_indices_,
        beta,
        C.description_,
        C.nnz_,
        C.csr_values_,
        C.csr_pointers_,
        C.csr_indices_,
        out.description_,
        out.csr_values_.data_ptr<double>(),
        out.csr_pointers_.data_ptr<int>(),
        out.csr_indices_.data_ptr<int>(),
        gemm2Info_,
        buffer_));
    return out;
  }
};

// Specializacion for `A @ B` operation for float values with cuSparse
template<> struct CusparseMatrixMultiplyOp<float> {
  csrgemm2Info_t gemm2Info_;

  CusparseMatrixMultiplyOp() {
    TORCH_CUDASPARSE_CHECK(hipsparseCreateCsrgemm2Info(&gemm2Info_));

  }
  ~CusparseMatrixMultiplyOp() {
    hipsparseDestroyCsrgemm2Info(gemm2Info_);
  }
  csrOutput operator()(
      const ScsrMatrixRef& lhs,
      const ScsrMatrixRef& rhs,
      Tensor &output_values,
      Tensor &output_indices) {
    float alpha = 1.0;
    ScsrMatrixRef empty;
    return Sgemm2(lhs, rhs, empty, &alpha, nullptr, output_values, output_indices);
  }

  csrOutput Sgemm2(
      const ScsrMatrixRef& A,
      const ScsrMatrixRef& B,
      const ScsrMatrixRef& C,
      const float* alpha,
      const float* beta,
      Tensor &output_values,
      Tensor &output_indices) {
    void* buffer_{nullptr};
    hipsparseHandle_t cusparseHandle_ = at::cuda::getCurrentCUDASparseHandle();
    TORCH_CUDASPARSE_CHECK(hipsparseSetPointerMode(cusparseHandle_, HIPSPARSE_POINTER_MODE_HOST));

    csrOutput out({A.size(0), B.size(1)});

    int innerSize = confirm_mult_size(A.size_, B.size_);

    out.csr_pointers_ = at::empty({out.size(0) + 1}, output_indices.options().dtype(kInt));

    // Compute needed buffer size
    size_t new_bubber_sz;
    TORCH_CUDASPARSE_CHECK(hipsparseScsrgemm2_bufferSizeExt(
        cusparseHandle_,
        out.size(0),
        out.size(1),
        innerSize,
        alpha,
        A.description_,
        A.nnz_,
        A.csr_pointers_,
        A.csr_indices_,
        B.description_,
        B.nnz_,
        B.csr_pointers_,
        B.csr_indices_,
        beta,
        C.description_,
        C.nnz_,
        C.csr_pointers_,
        C.csr_indices_,
        gemm2Info_,
        &new_bubber_sz));

    auto& allocator = *::c10::cuda::CUDACachingAllocator::get();
    at::DataPtr data_ptr = allocator.allocate(new_bubber_sz);
    buffer_ = data_ptr.get();

    // Find the resulting non-zero pattern.
    TORCH_CUDASPARSE_CHECK(hipsparseXcsrgemm2Nnz(
        cusparseHandle_,
        out.size(0),
        out.size(1),
        innerSize,
        A.description_,
        A.nnz_,
        A.csr_pointers_,
        A.csr_indices_,
        B.description_,
        B.nnz_,
        B.csr_pointers_,
        B.csr_indices_,
        C.description_,
        C.nnz_,
        C.csr_pointers_,
        C.csr_indices_,
        out.description_,
        out.csr_pointers_.data_ptr<int>(),
        &out.nnz_,
        gemm2Info_,
        buffer_));

    out.csr_indices_ = at::empty({out.nnz_}, output_indices.options().dtype(kInt));
    out.csr_values_ = at::empty({out.nnz_}, output_values.options());

    // Perform the gemm2 operation for doubles
    // out = alpha ∗ A ∗ B + beta ∗ C
    TORCH_CUDASPARSE_CHECK(hipsparseScsrgemm2(
        cusparseHandle_,
        out.size(0),
        out.size(1),
        innerSize,
        alpha,
        A.description_,
        A.nnz_,
        A.csr_values_,
        A.csr_pointers_,
        A.csr_indices_,
        B.description_,
        B.nnz_,
        B.csr_values_,
        B.csr_pointers_,
        B.csr_indices_,
        beta,
        C.description_,
        C.nnz_,
        C.csr_values_,
        C.csr_pointers_,
        C.csr_indices_,
        out.description_,
        out.csr_values_.data_ptr<float>(),
        out.csr_pointers_.data_ptr<int>(),
        out.csr_indices_.data_ptr<int>(),
        gemm2Info_,
        buffer_));
    return out;
  }
};



#endif // IS_CUSPARSE11_AVAILABLE()

template <typename scalar_t>
void sparse_sparse_matmul_cuda_kernel(
    Tensor& result,
    const Tensor& mat1,
    const Tensor& mat2) {

  static_assert(
    std::is_same<c10::Half, scalar_t>::value ||
        std::is_same<c10::BFloat16, scalar_t>::value ||
        std::is_same<float, scalar_t>::value ||
        std::is_same<double, scalar_t>::value ||
        std::is_same<c10::complex<float>, scalar_t>::value ||
        std::is_same<c10::complex<double>, scalar_t>::value,
    "sparse_sparse_matmul_cuda_kernel only supports data type of half, bfloat16, float, double and complex float, double.");

  // older versions of cusparse on Windows segfault for complex128 dtype
#if defined(_WIN32) && defined(CUSPARSE_VERSION) && CUSPARSE_VERSION < 11400
  TORCH_CHECK(
      !(mat1.scalar_type() == ScalarType::ComplexDouble),
      "Sparse multiplication with complex128 dtype inputs is not supported with current CUDA version. Please upgrade to CUDA Toolkit 11.2.1+");
#endif

  Tensor mat1_indices_ = mat1._indices().contiguous();
  Tensor mat1_values = mat1._values().contiguous();

  Tensor mat1_row_indices = mat1_indices_.select(0, 0);
  Tensor mat1_col_indices = mat1_indices_.select(0, 1);

  Tensor mat1_indptr = _to_csr_int(mat1_row_indices, mat1.size(0), mat1._nnz());

  Tensor mat1_indices = at::empty(
      {mat1_col_indices.size(0)}, mat1_col_indices.options().dtype(kInt));

  mat1_indices.copy_(mat1_col_indices);

  Tensor mat2_indices_ = mat2._indices().contiguous();
  Tensor mat2_values = mat2._values().contiguous();
  Tensor mat2_row_indices = mat2_indices_.select(0, 0);
  Tensor mat2_col_indices = mat2_indices_.select(0, 1);

  Tensor mat2_indptr = _to_csr_int(mat2_row_indices, mat2.size(0), mat2._nnz());
  Tensor mat2_indices = at::empty({mat2_col_indices.size(0)}, mat2_col_indices.options().dtype(kInt));
  mat2_indices.copy_(mat2_col_indices);

  auto m = mat1.size(0);
  auto k1 = mat1.size(1);

  auto k2 = mat2.size(0);
  auto n = mat2.size(1);
  TORCH_CHECK((m <= INT_MAX) && (n <= INT_MAX) && (k1 <= INT_MAX),
    "At the moment, hipsparseDcsrgemm2 only supports m, n, k, nnz with the bound [val] <= ", INT_MAX, ".",
    "If you need this, please file an issue on GitHub."
  );
  auto output_indices = result._indices();
  auto output_values = result._values();

  if ((k1 == 0 && k2 == 0) || (n == 0 && m == 0)) {
    output_indices.zero_();
    output_values.zero_();
    return;
  }

  csrMatrixRef<scalar_t> csr_mat1(
      mat1_indices.data_ptr<int>(),
      mat1_indptr.data_ptr<int>(),
      mat1_values.data_ptr<scalar_t>(),
      (int)mat1._nnz(),
      {(int)mat1.size(0), (int)mat1.size(1)});

  csrMatrixRef<scalar_t> csr_mat2(
      mat2_indices.data_ptr<int>(),
      mat2_indptr.data_ptr<int>(),
      mat2_values.data_ptr<scalar_t>(),
      (int)mat2._nnz(),
      {(int)mat2.size(0), (int)mat2.size(1)});

  // Sparse matrix multiplication
  CusparseMatrixMultiplyOp<scalar_t> op;
  csrOutput csr_output = op(csr_mat1, csr_mat2, output_values, output_indices);
  auto nnz = csr_output.nnz_;

  output_values.set_(csr_output.csr_values_);
  output_indices.resize_({2, nnz});
  auto output_indices_accessor = output_indices.packed_accessor64<int64_t, 2>();

  auto csr_output_pointers_accessor =
      csr_output.csr_pointers_.packed_accessor64<int, 1>();

  auto csr_output_ind_accessor =
      csr_output.csr_indices_.packed_accessor64<int, 1>();

  auto major_dim = result.size(0);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  at::cuda::ThrustAllocator allocator;
  auto policy = thrust::cuda::par(allocator).on(stream);

  // Filling the COO row indices
  thrust::for_each(
      policy,
      thrust::make_counting_iterator(int64_t(0)),
      thrust::make_counting_iterator(int64_t(major_dim)),
      [output_indices_accessor,
       csr_output_pointers_accessor,
       major_dim,
       nnz] __device__(int64_t i) {
        auto Ap = csr_output_pointers_accessor.data();
        int64_t* indices_row = output_indices_accessor[0].data();

        for (int jj = Ap[i];  jj < Ap[i + 1]; jj++) {
          indices_row[jj] = i;
        }
      });

  // Filling the COO column indices
  thrust::for_each(
    policy,
    thrust::make_counting_iterator(int64_t(0)),
    thrust::make_counting_iterator(int64_t(csr_output.nnz_)),
    [output_indices_accessor,
      csr_output_pointers_accessor,
      csr_output_ind_accessor,
      major_dim,
      nnz] __device__(int64_t i) {
      int64_t* indices_col = output_indices_accessor[1].data();
      indices_col[i] = csr_output_ind_accessor[i];
    });
}

} // end anonymous namespace

Tensor sparse_sparse_matmul_cuda(const Tensor& mat1_, const Tensor& mat2_) {
  TORCH_INTERNAL_ASSERT(mat1_.is_sparse());
  TORCH_INTERNAL_ASSERT(mat2_.is_sparse());
  TORCH_CHECK(mat1_.dim() == 2);
  TORCH_CHECK(mat2_.dim() == 2);
  TORCH_CHECK(mat1_.dense_dim() == 0, "sparse_mm: scalar values expected, mat1 got ", mat1_.dense_dim(), "D values");
  TORCH_CHECK(mat2_.dense_dim() == 0, "sparse_mm: scalar values expected, mat2 got ", mat2_.dense_dim(), "D values");

  TORCH_CHECK(
      mat1_.size(1) == mat2_.size(0), "mat1 and mat2 shapes cannot be multiplied (",
      mat1_.size(0), "x", mat1_.size(1), " and ", mat2_.size(0), "x", mat2_.size(1), ")");

  TORCH_CHECK(mat1_.scalar_type() == mat2_.scalar_type(),
           "mat1 dtype ", mat1_.scalar_type(), " does not match mat2 dtype ", mat2_.scalar_type());

  auto output = at::native::empty_like(mat1_);
  output.sparse_resize_and_clear_({mat1_.size(0), mat2_.size(1)}, mat1_.sparse_dim(), 0);

#if IS_CUSPARSE11_AVAILABLE()
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(kHalf, kBFloat16, mat1_.scalar_type(), "sparse_matmul", [&] {
    sparse_sparse_matmul_cuda_kernel<scalar_t>(output, mat1_.coalesce(), mat2_.coalesce());
  });
#else
  AT_DISPATCH_FLOATING_TYPES(mat1_.scalar_type(), "sparse_matmul", [&] {
    sparse_sparse_matmul_cuda_kernel<scalar_t>(output, mat1_.coalesce(), mat2_.coalesce());
  });
#endif
  return output;
}

} // namespace native
} // namespace at
